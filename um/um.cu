#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

void myCudaPrefetch(float *x, int s)
{
  hipMemPrefetchAsync(x, s, 0);
}



// CUDA kernel to add elements of two arrays
__global__
void Myadd(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}
 
int main(void)
{
  int N = 1<<10;
  float *x, *y;

  hipStream_t stream;
  hipStreamCreate(&stream);
 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // myCudaPrefetch(x, N*sizeof(float));
  // hipMemPrefetchAsync(x, N*sizeof(float), 0);
  // hipMemPrefetchAsync(y, N*sizeof(float), 0);
 
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
 
  // Launch kernel on 1M elements on the GPU
  int blockSize = 128;
  int numBlocks = (N + blockSize - 1) / blockSize;
  Myadd<<<numBlocks, blockSize>>>(N, x, y);
 
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
 
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  hipFree(x);
  hipFree(y);
 
  return 0;
}
